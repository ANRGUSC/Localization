#include "hip/hip_runtime.h"
#include "main.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

// Use a multiple of 4
#define MAX_THREADS 8

void kernelDriver();


__global__ void cuda_gaussian(double *out_likelihood, double *in_obs, double *in_Tx, double *in_Ty, double *in_eta, 
								double *in_sigma, int *in_lmax, int *in_wmax, int *in_totthreads) 
{
	int thrId = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (thrId >= *in_totthreads)
		return;

	// printf("Hello World from GPU!\t%lu\n", thrId); return;

	int inmr = thrId, idnr = ((*in_totthreads)/ (*in_lmax));
	int jnmr = (inmr%idnr), jdnr = (idnr / (*in_wmax));
	int pnmr = (jnmr%jdnr), pdnr = (jdnr / (*in_lmax));
	int p = pnmr / pdnr;
	int q = pnmr % pdnr;

	double obs_r[3];
	double likelihood = 1.0;
	int iter;
	int obsr_iter;

	for (iter = 0, obsr_iter = 0; iter < 3; iter++)
	{
		double dist = sqrt((pow(((in_Tx[iter]) - p), 2)) +
			(pow(((in_Ty[iter]) - q), 2)));

		if (dist == 0) continue;
		obs_r[obsr_iter++] = -10 * (*in_eta) * log(dist);
	}

	for (iter = 0; iter < obsr_iter; iter++)
	{
		double temp_likelihood = (1.0 / ((sqrt(2 * M_PI)) * (*in_sigma))) *
									(exp((-pow(in_obs[iter] - obs_r[iter], 2.0)) /
									(2 * pow((*in_sigma), 2.0))));
		likelihood *= temp_likelihood;
	}
	out_likelihood[thrId] =  likelihood;
	return;
}

 void kernelDriver(int args_numThr) 
 {	
		double *kern_inputobs;
		double *kern_inputTx;
		double *kern_inputTy;
		double *kern_inputeta;
		double *kern_inputsigma;
		int *kern_inputlmax;
		int *kern_inputwmax;
		int *kern_inputtotthreads;

		double *kern_outputArr;
		hipError_t error;

		// Malloc Session
		if ((error = hipMalloc((void**)&kern_outputArr, (config->l * config->l * config->w * config->w) * sizeof(double))) != hipSuccess)
		{
			std::cout << "hipMalloc returned error for kernel outputArr" << std::endl;
		}

		if ((error = hipMalloc((void**)&kern_inputobs, 3 * sizeof(double))) != hipSuccess)
		{
			std::cout << "hipMalloc returned error for kernel inputObs" << std::endl;
		}

		if ((error = hipMalloc((void**)&kern_inputTx, 3 * sizeof(double))) != hipSuccess)
		{
			std::cout << "hipMalloc returned error for kernel inputTx" << std::endl;
		}

		if ((error = hipMalloc((void**)&kern_inputTy, 3 * sizeof(double))) != hipSuccess)
		{
			std::cout << "hipMalloc returned error for kernel inputTy" << std::endl;
		}

		if ((error = hipMalloc((void**)&kern_inputeta, sizeof(double))) != hipSuccess)
		{
			std::cout << "hipMalloc returned error for kernel inputeta" << std::endl;
		}

		if ((error = hipMalloc((void**)&kern_inputsigma, sizeof(double))) != hipSuccess)
		{
			std::cout << "hipMalloc returned error for kernel inputsigma" << std::endl;
		}

		if ((error = hipMalloc((void**)&kern_inputlmax, sizeof(int))) != hipSuccess)
		{
			std::cout << "hipMalloc returned error for kernel inputlmax" << std::endl;
		}

		if ((error = hipMalloc((void**)&kern_inputwmax, sizeof(int))) != hipSuccess)
		{
			std::cout << "hipMalloc returned error for kernel inputwmax" << std::endl;
		}

		if ((error = hipMalloc((void**)&kern_inputtotthreads, sizeof(int))) != hipSuccess)
		{
			std::cout << "hipMalloc returned error for kernel inputtotthreads" << std::endl;
		}


		// Memcpy Session
		double* obs_arr = &config->obs[0];
		error = hipMemcpy(kern_inputobs, obs_arr, 3 * sizeof(double), hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			std::cout << "hipMemcpy returned error for copying kern_inputobs" << std::endl;
		}

		double* Tx_arr = &config->Tx[0];
		error = hipMemcpy(kern_inputTx, Tx_arr, 3 * sizeof(double), hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			std::cout << "hipMemcpy returned error for copying kern_inputTx" << std::endl;
		}

		double* Ty_arr = &config->Ty[0];
		error = hipMemcpy(kern_inputTy, Ty_arr, 3 * sizeof(double), hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			std::cout << "hipMemcpy returned error for copying kern_inputTy" << std::endl;
		}

		error = hipMemcpy(kern_inputeta, &config->eta, sizeof(double), hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			std::cout << "hipMemcpy returned error for copying kern_inputeta" << std::endl;
		}

		error = hipMemcpy(kern_inputsigma, &config->sigma, sizeof(double), hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			std::cout << "hipMemcpy returned error for copying kern_inputsigma" << std::endl;
		}

		error = hipMemcpy(kern_inputlmax, &config->l, sizeof(int), hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			std::cout << "hipMemcpy returned error for copying kern_inputlmax" << std::endl;
		}

		error = hipMemcpy(kern_inputwmax, &config->w, sizeof(int), hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			std::cout << "hipMemcpy returned error for copying kern_inputwmax" << std::endl;
		}

		int totthreads = config->l * config->w * config->l * config->w;
		error = hipMemcpy(kern_inputtotthreads, &totthreads, sizeof(int), hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			std::cout << "hipMemcpy returned error for copying kern_inputtotthreads" << std::endl;
		}

		unsigned int numBlocks = ((unsigned int)floor((config->l * config->l * config->w * config->w) / args_numThr) + 1) * 4;
		int numThreads = args_numThr/4;
		hipDeviceSynchronize();
		cuda_gaussian << <numBlocks, numThreads >> > (kern_outputArr, kern_inputobs, kern_inputTx, kern_inputTy, kern_inputeta,
														kern_inputsigma, kern_inputlmax, kern_inputwmax, kern_inputtotthreads);
		hipDeviceSynchronize();

		hipMemcpy(likelihoodmatrix, kern_outputArr, (config->l * config->l * config->w * config->w) * sizeof(double), hipMemcpyDeviceToHost);

}
