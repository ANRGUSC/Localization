#include "hip/hip_runtime.h"
#include "main.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

// #define __TESTMODE__

#ifndef __TESTMODE__
#define IN_MAT_ROWS		3000
#define NUM_BLOCKS		10
#else
#define IN_MAT_ROWS		9
#define NUM_BLOCKS		3
#endif

#define IN_MAT_COLS			3
#define OUT_MAT_COLS		1
#define OUT_MAT_ROWS		IN_MAT_ROWS
#define NUM_THRDS_PER_BLK	((IN_MAT_ROWS/NUM_BLOCKS) * IN_MAT_COLS)


void kernelDriver(size_t);


__global__ void cuda_hello() {
	int thrId_Act = blockIdx.x * blockDim.x + threadIdx.x;
	printf("Hello World from GPU!\t%d\t%lu\n", blockIdx.x, thrId_Act);
}


__global__ void cuda_gaussian(double *outArr, double *inArr) {
	int thrId_Blk = threadIdx.x;
	int thrId_Act = blockIdx.x * blockDim.x + threadIdx.x;
	int opmode = thrId_Blk % IN_MAT_COLS;
	double comp_a_factor = sqrt(2 * M_PI);
	__device__ __shared__ double gaussianIntrComp_t[NUM_THRDS_PER_BLK];

	switch (opmode)
	{
		case 0:
		{
			gaussianIntrComp_t[thrId_Blk] = -pow((inArr[thrId_Act] - inArr[thrId_Act + 1]), 2.0);
			break;
		}

		case 1:
		{
			gaussianIntrComp_t[thrId_Blk] = 2 * pow(inArr[thrId_Act + 1], 2.0);
			break;
		}

		case 2:
		{
			gaussianIntrComp_t[thrId_Blk] = 1.0 / (comp_a_factor * inArr[thrId_Act]);
			break;
		}

		default:
			break;
	}

	__syncthreads();
	if (opmode == 2)
	{
		double comp_d;
		comp_d = pow(M_E, (gaussianIntrComp_t[thrId_Blk - 2] / gaussianIntrComp_t[thrId_Blk - 1]));
		outArr[thrId_Act / IN_MAT_COLS] = gaussianIntrComp_t[thrId_Blk] * comp_d;
	}
	return;
}

__global__ void simplecudaAdd(double* out, double* in)
{
	printf("Hello World from GPU!\n");
	for (int i = 0; i < (IN_MAT_ROWS * IN_MAT_COLS); i++)
	{
		printf("%f\n", in[i]);
	}
	for (int i = 0; i < 9; i++)
	{
		out[i] = 10.0;
		printf(" %d\t\t%d\t\t%d\t\t%d\n", (blockIdx.x * blockDim.x + threadIdx.x), blockIdx.x, blockDim.x, threadIdx.x);
	}

}

 void kernelDriver(size_t curIter) 
 {
	 size_t outer_var = curIter;
	//for (size_t outer_var = 0; outer_var < kernelnumRows; outer_var+=3000)
	//{
		double *inputGPUXferArr = (double*) new double[IN_MAT_ROWS * IN_MAT_COLS];
		double *outputGPUXferArr = (double*) new double[OUT_MAT_ROWS * OUT_MAT_COLS];
		double *kern_inputArr;
		double *kern_outputArr;
		hipError_t error;
		double *kernOutputArrPtr = &kudaOutMatrix[0];

		if ((error = hipMalloc((void**)&kern_outputArr, OUT_MAT_ROWS * OUT_MAT_COLS * sizeof(double))) != hipSuccess)
		{
			std::cout << "hipMalloc returned error for kernel outputArr" << std::endl;
		}

		if ((error = hipMalloc((void**)&kern_inputArr, IN_MAT_ROWS * IN_MAT_COLS * sizeof(double))) != hipSuccess)
		{
			std::cout << "hipMalloc returned error for kernel inputArr" << std::endl;
		}

		for (size_t i = outer_var, counter_i = 0; i < (outer_var + 3000); i++)
		{
			for (size_t j = 0; j < IN_MAT_COLS; j++, counter_i++)
				inputGPUXferArr[counter_i] = kudaTransferKernelMatrix[i][j];
		}

		error = hipMemcpy(kern_inputArr, inputGPUXferArr, IN_MAT_ROWS * IN_MAT_COLS * sizeof(double), hipMemcpyHostToDevice);
		if (error != hipSuccess)
		{
			std::cout << "hipMemcpy returned error for copying kernel size" << std::endl;
		}

		hipDeviceSynchronize();

		cuda_gaussian << <NUM_BLOCKS, NUM_THRDS_PER_BLK >> > (kern_outputArr, kern_inputArr);
		hipMemcpy(kernOutputArrPtr + outer_var, kern_outputArr, OUT_MAT_ROWS * sizeof(double), hipMemcpyDeviceToHost);
	//}
}